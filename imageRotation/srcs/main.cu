#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
/********************/
__global__ void kernel(cv::Mat *d_ori, cv::Mat *d_dest, int rows, int cols)
{
	int index_x = blockIdx.x * blockDim.x + threadIdx.x; // access thread # within all grids
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int grid_width = gridDim.x * blockDim.x; // access # of thread
	int index = index_y * grid_width + index_x; // access thread # within all grids
	int r = blockIdx.y * gridDim.x + blockIdx.x; // access block # within all grids

	int x = (index * cols - index) / (rows * cols - 1);
	int y = ((rows - 1) * index) / (rows * cols - 1);

	//printf("[%d, %d]\n", index_x, index_y);
	printf("[%d, %d, %d ||| %d, %d, %d] --> %d, %d ||| [%d] --> %d --> %d ||| %d !!! %d, %d\n", blockIdx.x, blockDim.x, threadIdx.x, blockIdx.y, blockDim.y, threadIdx.y, index_x, index_y, gridDim.x, grid_width, index, r, x, y);
	
	return ;
}

void			cudaTest(cv::Mat *h_ori, cv::Mat *h_dest)
{
	cv::Mat		*d_ori, *d_dest;
	int			rows = h_ori->rows;
	int			cols = h_ori->cols;
	int			pixels = rows * cols;
	size_t		blockSize, gridSize;

	hipMalloc((void**)&d_ori, sizeof(cv::Mat));
	hipMalloc((void**)&d_dest, sizeof(cv::Mat));

	// blockSize = 1;
	// gridSize = pixels / blockSize;
	// if (pixels % blockSize)
	// 	++gridSize;

	hipMemcpy(d_ori, h_ori, sizeof(cv::Mat), hipMemcpyHostToDevice);
	hipMemcpy(d_dest, h_dest, sizeof(cv::Mat), hipMemcpyHostToDevice);
	// kernel<<< gridSize, blockSize >>>(d_dest, rows, cols);
	kernel<<< rows, cols >>>(d_ori, d_dest, rows, cols);
	hipMemcpy(h_dest, d_dest, sizeof(cv::Mat), hipMemcpyDeviceToHost);
	hipFree(d_ori);
	hipFree(d_dest);
	return ;
}
/********************/

int				main(int ac, char **av)
{
	cv::Mat		*h_originImg, *h_destImg;
	char		*originName, *destName;
	int			degreeRot;

	if (ac != 4) {
		std::cout << "usage: ./cudaRotate src_name dest_name degree_rot" << std::endl;
		return (1);
	}
	try {
		degreeRot = std::stoi(av[3]);
	} catch (std::exception & e) {
		std::cout << "error: " << e.what() << std::endl;
		return (2);
	}
	originName = av[1];
	destName = av[2];
	h_originImg = new cv::Mat(2, 2, CV_8SC1);/////////////
	//*h_originImg = cv::imread(originName, CV_8SC1);
	h_destImg = new cv::Mat();
	cudaTest(h_originImg, h_destImg);//////////////
	//h_originImg->copyTo(*h_destImg);////////////////
	try {
		cv::imwrite(destName, *h_destImg);
	} catch (std::exception & e) {
		std::cout << "error: " << e.what() << std::endl;
		return (3);
	}
	delete h_originImg;
	delete h_destImg;
	return (0);
}
