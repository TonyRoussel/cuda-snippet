#include "hip/hip_runtime.h"
#include "util/cuPrintf.cu"
#include <stdio.h>

__global__ void device_greetings(void)
{
  cuPrintf("Hello, world from the device!\n");
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // initialize cuPrintf
  cudaPrintfInit();

  // launch a kernel with a single thread to greet from the device
  device_greetings<<<10,64>>>();

  // display the device's greeting
  cudaPrintfDisplay();
  
  // clean up after cuPrintf
  cudaPrintfEnd();

  return 0;
}